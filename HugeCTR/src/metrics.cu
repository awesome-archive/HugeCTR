#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cub/hipcub/hipcub.hpp>
#include <diagnose.hpp>
#include <metrics.hpp>
#include <utils.cuh>

namespace HugeCTR {

namespace metrics {

namespace {

const float eps = 1e-32;

__global__ void unique_flag_kernel(const float* data, char* flag, int num_elems) {
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_elems - 1; gid += blockDim.x * gridDim.x) {
    float lhs = data[gid];
    float rhs = data[gid + 1];
    // assume the elements are in descending order
    flag[gid] = ((lhs - rhs) > eps) ? 1 : 0;
  }
  if (gid_base == 0) {
    flag[num_elems - 1] = 1;
  }
}

__global__ void unique_index_kernel(const char* flag, const int* flag_inc_sum, int* unique_index,
                                    int num_elems) {
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_elems; gid += blockDim.x * gridDim.x) {
    if (flag[gid] == 1) {
      int id = flag_inc_sum[gid] - 1;
      unique_index[id] = gid;
    }
  }
}

__global__ void create_fpr_kernel(float* tpr, const int* unique_index, float* fpr, int num_selected,
                                  int num_total) {
  float pos_cnt = tpr[num_selected - 1];
  float neg_cnt = num_total - pos_cnt;
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_selected; gid += blockDim.x * gridDim.x) {
    float tp = tpr[gid];
    fpr[gid] = (1.0f + unique_index[gid] - tp) / neg_cnt;
    tpr[gid] = tp / pos_cnt;
  }
}

__global__ void trapz_kernel(float* y, float* x, float* auc, int num_selected) {
  __shared__ float s_auc;
  s_auc = 0.0f;
  __syncthreads();
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_selected - 1; gid += blockDim.x * gridDim.x) {
    float a = x[gid];
    float b = x[gid + 1];
    float fa = y[gid];
    float fb = y[gid + 1];
    float area = (b - a) * (fa + fb) / 2.0f;
    if (gid == 0) {
      area += (a * fa / 2.0f);
    }
    atomicAdd(&s_auc, area);
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    atomicAdd(auc, s_auc);
  }
}

/* __global__ void half2float_kernel(float* y, const __half* x, int num_elems) {
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_elems; gid += blockDim.x * gridDim.x) {
    y[gid] = __half2float(x[gid]);
  }
} */

__global__ void copy_all_kernel(float* y_pred, float* y_label, const __half* x_pred,
                                const float* x_label, int num_elems) {
  int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
  for (int gid = gid_base; gid < num_elems; gid += blockDim.x * gridDim.x) {
    float pred_val = __half2float(x_pred[gid]);
    float label_val = x_label[gid];
    y_pred[gid] = pred_val;
    y_label[gid] = label_val;
  }
}

template <typename SrcType>
void copy_pred(float* y, SrcType* x, int num_elems, int num_sms, hipStream_t stream);

template <>
void copy_pred<float>(float* y, float* x, int num_elems, int num_sms, hipStream_t stream) {
  CK_CUDA_THROW_(
      hipMemcpyAsync(y, x, num_elems * sizeof(float), hipMemcpyDeviceToDevice, stream));
}

/* template <>
void copy_pred<__half>(float* y, __half* x, int num_elems, int num_sms, hipStream_t stream) {
  dim3 grid(num_sms * 2, 1, 1);
  dim3 block(1024, 1, 1);
  half2float_kernel<<<grid, block, 0, stream>>>(y, x, num_elems);
} */

template <typename PredType>
void copy_all(float* y_pred, float* y_label, PredType* x_pred, float* x_label, int num_elems,
              int num_sms, hipStream_t stream);

template <>
void copy_all<float>(float* y_pred, float* y_label, float* x_pred, float* x_label, int num_elems,
                     int num_sms, hipStream_t stream) {
  copy_pred<float>(y_pred, x_pred, num_elems, num_sms, stream);
  CK_CUDA_THROW_(hipMemcpyAsync(y_label, x_label, num_elems * sizeof(float),
                                 hipMemcpyDeviceToDevice, stream));
}

template <>
void copy_all<__half>(float* y_pred, float* y_label, __half* x_pred, float* x_label, int num_elems,
                      int num_sms, hipStream_t stream) {
  dim3 grid(num_sms * 2, 1, 1);
  dim3 block(1024, 1, 1);
  copy_all_kernel<<<grid, block, 0, stream>>>(y_pred, y_label, x_pred, x_label, num_elems);
}

}  // namespace

std::unique_ptr<Metric> Metric::Create(const Type type, bool use_mixed_precision,
                                       int batch_size_eval, int n_batches,
                                       const std::shared_ptr<ResourceManager>& resource_manager) {
  std::unique_ptr<Metric> ret;
  switch (type) {
    case Type::AUC:
      if (use_mixed_precision) {
        ret.reset(new AUC<__half>(batch_size_eval, n_batches, resource_manager));
      } else {
        ret.reset(new AUC<float>(batch_size_eval, n_batches, resource_manager));
      }
      break;
    case Type::AverageLoss:
      ret.reset(new AverageLoss<float>(resource_manager));
      break;
  }
  return ret;
}

Metric::Metric() : num_procs_(1), pid_(0), current_batch_size_(0) {
#ifdef ENABLE_MPI
  CK_MPI_THROW_(MPI_Comm_rank(MPI_COMM_WORLD, &pid_));
  CK_MPI_THROW_(MPI_Comm_size(MPI_COMM_WORLD, &num_procs_));
#endif
}
Metric::~Metric() {}

template <typename T>
AverageLoss<T>::AverageLoss(const std::shared_ptr<ResourceManager>& resource_manager)
    : Metric(),
      resource_manager_(resource_manager),
      loss_local_(std::vector<float>(resource_manager->get_local_gpu_count(), 0.0f)),
      loss_global_(0.0f),
      n_batches_(0) {}

template <typename T>
AverageLoss<T>::~AverageLoss() {}

template <typename T>
void AverageLoss<T>::local_reduce(int local_gpu_id, RawMetricMap raw_metrics) {
  float loss_host = 0.0f;
  Tensor2<T> loss_tensor = Tensor2<T>::stretch_from(raw_metrics[RawType::Loss]);
  CudaDeviceContext context(resource_manager_->get_local_gpu(local_gpu_id)->get_device_id());
  CK_CUDA_THROW_(
      hipMemcpy(&loss_host, loss_tensor.get_ptr(), sizeof(float), hipMemcpyDeviceToHost));
  loss_local_[local_gpu_id] = loss_host;
}

template <typename T>
void AverageLoss<T>::global_reduce(int n_nets) {
  float loss_inter = 0.0f;
  for (auto& loss_local : loss_local_) {
    loss_inter += loss_local;
  }

#ifdef ENABLE_MPI
  if (num_procs_ > 1) {
    float loss_reduced = 0.0f;
    CK_MPI_THROW_(MPI_Reduce(&loss_inter, &loss_reduced, 1, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD));
    loss_inter = loss_reduced;
  }
#endif
  loss_global_ += loss_inter / n_nets / num_procs_;
  n_batches_++;
}

template <typename T>
float AverageLoss<T>::finalize_metric() {
  float ret = 0.0f;
  if (pid_ == 0) {
    if (n_batches_) {
      ret = loss_global_ / n_batches_;
    }
  }
#ifdef ENABLE_MPI
  CK_MPI_THROW_(MPI_Barrier(MPI_COMM_WORLD));
  CK_MPI_THROW_(MPI_Bcast(&ret, 1, MPI_FLOAT, 0, MPI_COMM_WORLD));
#endif

  loss_global_ = 0.0f;
  for (auto& loss_local : loss_local_) {
    loss_local = 0.0f;
  }
  n_batches_ = 0;
  return ret;
}

template <typename T>
AUC<T>::AUC(int batch_size_per_gpu, int n_batches,
            const std::shared_ptr<ResourceManager>& resource_manager)
    : Metric(),
      resource_manager_(resource_manager),
      batch_size_per_gpu_(batch_size_per_gpu),
      n_batches_(n_batches),
      root_device_id_(resource_manager->get_local_gpu(0)->get_device_id()),
      num_gpus_(resource_manager->get_local_gpu_count()),
      offset_(0),
      temp0_(nullptr),
      temp1_(nullptr),
      temp2_(nullptr),
      temp3_(nullptr),
      workspace_(nullptr),
      temp_storage_bytes_(0) {
  int num_elems = batch_size_per_gpu_ * n_batches_ * num_gpus_;
#ifdef ENABLE_MPI
  if (num_procs_ > 1 && pid_ == 0) {
    num_elems *= num_procs_;
  }
#endif
  size_t buffer_size = num_elems * sizeof(float);

  CudaDeviceContext context(root_device_id_);
  CK_CUDA_THROW_(hipMallocManaged(&temp0_, buffer_size));
  CK_CUDA_THROW_(hipMallocManaged(&temp1_, buffer_size));
  CK_CUDA_THROW_(hipMallocManaged(&temp2_, buffer_size));
  CK_CUDA_THROW_(hipMallocManaged(&temp3_, buffer_size));

  size_t new_temp_storage_bytes = 0;

  CK_CUDA_THROW_(hipcub::DeviceRadixSort::SortPairsDescending(nullptr, new_temp_storage_bytes,
                                                           d_pred(), d_pred_sort(), d_label(),
                                                           d_label_sort(), num_elems, 0));
  set_max_temp_storage_bytes(new_temp_storage_bytes);

  CK_CUDA_THROW_(hipcub::DeviceScan::InclusiveSum(nullptr, new_temp_storage_bytes, d_label_sort(),
                                               d_label(), num_elems));
  set_max_temp_storage_bytes(new_temp_storage_bytes);

  char* dummy_d_flags = nullptr;
  int* dummy_d_num_selected_out = nullptr;
  CK_CUDA_THROW_(hipcub::DeviceSelect::Flagged(nullptr, new_temp_storage_bytes, d_label(),
                                            dummy_d_flags, d_label_sort(), dummy_d_num_selected_out,
                                            num_elems));
  set_max_temp_storage_bytes(new_temp_storage_bytes);

  set_max_temp_storage_bytes(buffer_size);

  size_t flag_size = num_elems * sizeof(char);
  size_t num_size = sizeof(int);
  CK_CUDA_THROW_(hipMallocManaged(&workspace_, temp_storage_bytes_ + num_size + flag_size));

  for (int b = 0; b < n_batches_; b++) {
    for (int g = 0; g < num_gpus_; g++) {
      int offset = (g + b * num_gpus_) * batch_size_per_gpu_;
      size_t size = batch_size_per_gpu_ * sizeof(float);
      hipMemAdvise(d_pred() + offset, size, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
      hipMemAdvise(d_label() + offset, size, hipMemAdviseSetAccessedBy, g);
    }
  }
}

template <typename T>
AUC<T>::~AUC() {
  hipFree(temp0_);
  hipFree(temp1_);
  hipFree(temp2_);
  hipFree(temp3_);
  hipFree(workspace_);
}

template <typename T>
void AUC<T>::local_reduce(int local_gpu_id, RawMetricMap raw_metrics) {
  Tensor2<PredType> pred_tensor = Tensor2<PredType>::stretch_from(raw_metrics[RawType::Pred]);
  Tensor2<LabelType> label_tensor = Tensor2<LabelType>::stretch_from(raw_metrics[RawType::Label]);

  int device_id = resource_manager_->get_local_gpu(local_gpu_id)->get_device_id();
  CudaDeviceContext context(device_id);
  int num_active_gpu = 0;
  int r = 0;
  num_active_gpu_and_r(num_active_gpu, r);
  if (r) {
    num_active_gpu += 1;
  }

  if (device_id < num_active_gpu) {
    int num_elems = (r && device_id == num_active_gpu - 1) ? r : batch_size_per_gpu_;

    size_t offset = offset_ + batch_size_per_gpu_ * device_id;

    // TBD get_local_gpu
    copy_all<T>(d_pred() + offset, d_label() + offset, pred_tensor.get_ptr(),
                label_tensor.get_ptr(), num_elems,
                resource_manager_->get_local_gpu(local_gpu_id)->get_sm_count(),
                resource_manager_->get_local_gpu(local_gpu_id)->get_stream());
  }
}

template <typename T>
void AUC<T>::global_reduce(int n_nets) {
  int num_active_gpu = 0;
  int r = 0;
  num_active_gpu_and_r(num_active_gpu, r);
  offset_ += (batch_size_per_gpu_ * num_active_gpu + r);

#ifdef ENABLE_MPI
  if (num_procs_ > 1) {
    int cnt = offset_;
    CK_MPI_THROW_(MPI_Gather((pid_ == 0) ? MPI_IN_PLACE : d_pred(), cnt, MPI_FLOAT, d_pred(), cnt,
                             MPI_FLOAT, 0, MPI_COMM_WORLD));
    CK_MPI_THROW_(MPI_Gather((pid_ == 0) ? MPI_IN_PLACE : d_label(), cnt, MPI_FLOAT, d_label(), cnt,
                             MPI_FLOAT, 0, MPI_COMM_WORLD));
  }
#endif
}

template <typename T>
float AUC<T>::finalize_metric() {
  CudaDeviceContext context(root_device_id_);

  if (pid_ == 0) {
    for (int i = 0; i < num_gpus_; i++) {
      CudaDeviceContext context(resource_manager_->get_local_gpu(i)->get_device_id());
      CK_CUDA_THROW_(hipDeviceSynchronize());
    }

    int num_elems = offset_ * num_procs_;
    CK_CUDA_THROW_(hipcub::DeviceRadixSort::SortPairsDescending(workspace_, temp_storage_bytes_,
                                                             d_pred(), d_pred_sort(), d_label(),
                                                             d_label_sort(), num_elems, 0));
    int* d_num_selected_out = ((int*)workspace_) + temp_storage_bytes_ / sizeof(int);
    char* d_flag = ((char*)workspace_) + temp_storage_bytes_ + sizeof(int);

    dim3 grid(160, 1, 1);
    dim3 block(1024, 1, 1);
    unique_flag_kernel<<<grid, block>>>(d_pred_sort(), d_flag, num_elems);

    CK_CUDA_THROW_(hipcub::DeviceScan::InclusiveSum(workspace_, temp_storage_bytes_, d_label_sort(),
                                                 d_inc_sum(), num_elems));

    CK_CUDA_THROW_(hipcub::DeviceSelect::Flagged(workspace_, temp_storage_bytes_, d_inc_sum(), d_flag,
                                              tpr(), d_num_selected_out, num_elems));

    int num_selected = 0;
    CK_CUDA_THROW_(
        hipMemcpy(&num_selected, d_num_selected_out, sizeof(int), hipMemcpyDeviceToHost));

    CK_CUDA_THROW_(hipcub::DeviceScan::InclusiveSum(workspace_, temp_storage_bytes_, d_flag,
                                                 d_flag_inc_sum(), num_elems));

    unique_index_kernel<<<grid, block>>>(d_flag, d_flag_inc_sum(), d_unique_index(), num_elems);

    create_fpr_kernel<<<grid, block>>>(tpr(), d_unique_index(), fpr(), num_selected, num_elems);

    initialize_array<<<grid, block>>>(d_auc(), 1, 0.0f);

    trapz_kernel<<<grid, block>>>(tpr(), fpr(), d_auc(), num_selected);

    CK_CUDA_THROW_(hipDeviceSynchronize());
  }
  offset_ = 0;

#ifdef ENABLE_MPI
  CK_MPI_THROW_(MPI_Barrier(MPI_COMM_WORLD));
  CK_MPI_THROW_(MPI_Bcast(d_auc(), 1, MPI_FLOAT, 0, MPI_COMM_WORLD));
#endif

  return *d_auc();
}

template <typename T>
void AUC<T>::set_max_temp_storage_bytes(size_t& new_val) {
  temp_storage_bytes_ = (new_val > temp_storage_bytes_) ? new_val : temp_storage_bytes_;
  new_val = 0;
}

template <typename T>
void AUC<T>::num_active_gpu_and_r(int& num_active_gpu, int& r) {
  num_active_gpu = current_batch_size_ / (batch_size_per_gpu_ * num_procs_);
  r = current_batch_size_ % (batch_size_per_gpu_ * num_procs_);
}

template class AverageLoss<float>;
template class AUC<float>;
template class AUC<__half>;

}  // namespace metrics

}  // namespace HugeCTR
